#include "hip/hip_runtime.h"
//============================================================================
// Name        : HelloWorld.cpp
// Author      : Isaiah Spearman
// Version     :
// Copyright   : Your copyright notice
// Description : Hello World in C++, Ansi-style
//============================================================================
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <float.h>
#include <math.h>
#include <iostream>
#include "libarff/arff_parser.h"
#include "libarff/arff_data.h"
#include<hip/hip_runtime.h>

using namespace std;


__global__ void KNNGPU(int num_attributes, int num_instances, int k_num, float *dataset, int *class_arr, int *predictions)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < num_instances)
    {
    	float* dists = (float*)malloc(k_num * sizeof(float));
    	int* neighs = (int*)malloc(k_num * sizeof(int));
    	int* sdc_arr = (int*)malloc(k_num * sizeof(int));
    	int max_dist_index = 0;
		int min_dist_index = 0;

    	for(int a = 0; a < k_num; a++)
		{
			dists[a] = FLT_MAX;
		}

        for (int j = 0; j < num_instances; j++)
        {
            if (i == j) { continue; }

            float distance = 0;

            for (int k = 0; k < num_attributes; k++)
            {
                int diff = dataset[i * num_attributes + k] - dataset[j * num_attributes + k];
                distance += diff * diff;
            }

            for(int l = 0; l < k_num; l++)
			{
				if(dists[l] > dists[max_dist_index])
				{
					max_dist_index = l;
				}
			}
            distance = sqrt(distance);
			if(distance < dists[max_dist_index]) // select the closest one
			{
				dists[max_dist_index] = distance;
				neighs[max_dist_index] = j;
				sdc_arr[max_dist_index] = class_arr[j];
			}
        }

        int max_count = 0;
		int sdc;
		for(int b = 0; b < k_num; b++)
		{
			int count = 0;
			for(int c = 0; c < k_num; c++)
			{
				if(sdc_arr[b]==sdc_arr[c])
					count++;
			}
			if(count > max_count)
			{
				max_count = count;
				sdc = sdc_arr[b];
			}
		}

		predictions[i] = sdc;
    }
}

int* computeConfusionMatrix(int* predictions, ArffData* dataset)
{
    //printf("ccm\n");
    int* confusionMatrix = (int*)calloc(dataset->num_classes() * dataset->num_classes(), sizeof(int)); // matriz size numberClasses x numberClasses

    for(int i = 0; i < dataset->num_instances(); i++) // for each instance compare the true class and predicted class
    {
        //printf("%d\n", i);
        int trueClass = dataset->get_instance(i)->get(dataset->num_attributes() - 1)->operator int32();
        int predictedClass = predictions[i];

        confusionMatrix[trueClass*dataset->num_classes() + predictedClass]++;
    }

    return confusionMatrix;
}

float computeAccuracy(int* confusionMatrix, ArffData* dataset)
{
    int successfulPredictions = 0;

    for(int i = 0; i < dataset->num_classes(); i++)
    {
        successfulPredictions += confusionMatrix[i*dataset->num_classes() + i]; // elements in the diagnoal are correct predictions
    }

    return successfulPredictions / (float) dataset->num_instances();
}

int main(int argc, char *argv[]) {
	if (argc != 3) {
		cout << "Usage: ./main datasets/datasetFile.arff K" << endl;
		exit(0);
	}

	ArffParser parser(argv[1]);
	ArffData *dataset = parser.parse();

	struct timespec start, end;
	uint64_t diff;

    const int num_attributes = dataset->num_attributes() - 1;
    const int num_instances = dataset->num_instances();


    int* actual_classes_host;
    int* actual_classes_device;
    int* predictions_device;
    float* data_instances_device;
    float* data_instances_host;

    int* h_A = (int*)malloc(dataset->num_instances() * sizeof(int));
    hipMalloc((void**)&data_instances_device, num_attributes * num_instances * sizeof(float));
    hipMalloc((void**)&actual_classes_device, num_instances * sizeof(int));
    hipMalloc((void**)&predictions_device, num_instances * sizeof(int));

    hipHostMalloc((void**)&data_instances_host, num_attributes * num_instances * sizeof(float));
    hipHostMalloc((void**)&actual_classes_host, num_instances * sizeof(int));

    for (int i = 0; i < num_instances; i++)
    {
        actual_classes_host[i] = dataset->get_instance(i)->get(num_attributes)->operator int32();
        for (int j = 0; j < num_attributes; j++)
        {
            data_instances_host[i * num_attributes + j] = dataset->get_instance(i)->get(j)->operator float();
        }
    }

    clock_gettime(CLOCK_MONOTONIC_RAW, &start);

    hipMemcpy((void*)data_instances_device, (void*)data_instances_host, num_attributes * num_instances * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((void*)actual_classes_device, (void*)actual_classes_host, num_instances * sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 128;
    int gridSize = (num_instances + blockSize - 1) / blockSize;

    KNNGPU <<< blockSize, gridSize >>> (num_attributes, num_instances, stoi(argv[2]), data_instances_device, actual_classes_device, predictions_device);

    hipMemcpy((void*)h_A, (void*)predictions_device, num_instances * sizeof(int), hipMemcpyDeviceToHost);

    clock_gettime(CLOCK_MONOTONIC_RAW, &end);
	diff = (1000000000L * (end.tv_sec - start.tv_sec) + end.tv_nsec - start.tv_nsec) / 1e6;

	int* confusionMatrix = computeConfusionMatrix(h_A, dataset);
	float accuracy = computeAccuracy(confusionMatrix, dataset);
	printf("The %sNN classifier sequential for %lu instances required %llu ms CPU time, accuracy was %.4f\n", argv[2], dataset->num_instances(), (long long unsigned int) diff, accuracy);

	hipFree(data_instances_device);
	hipFree(predictions_device);
	hipFree(actual_classes_device);
	hipFree(actual_classes_host);
	hipFree(data_instances_host);
	hipFree(h_A);

	return 0;
}
